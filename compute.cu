#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "vector.h"
#include "compute.h"
#include "config.h"

#define BLOCK_SIZE 256

__global__ void computeForces(int n, vector3 *pos, vector3 *vel, double *mass, vector3 *force) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int j;
        vector3 f = {0.0, 0.0, 0.0};
        for (j = 0; j < n; j++) {
            if (i != j) {
                vector3 r = {pos[j].x - pos[i].x, pos[j].y - pos[i].y, pos[j].z - pos[i].z};
                vector3 r_mag = {r.x / (sqrt(r.x * r.x + r.y * r.y + r.z * r.z)),
                                 r.y / (sqrt(r.x * r.x + r.y * r.y + r.z * r.z)),
                                 r.z / (sqrt(r.x * r.x + r.y * r.y + r.z * r.z))};
                vector3 dist = {sqrt(r.x * r.x + r.y * r.y + r.z * r.z),
                                sqrt(r.x * r.x + r.y * r.y + r.z * r.z),
                                sqrt(r.x * r.x + r.y * r.y + r.z * r.z)};
                vector3 mag = {GRAV_CONSTANT * mass[i] * mass[j] / (dist.x * dist.x * dist.x),
                               GRAV_CONSTANT * mass[i] * mass[j] / (dist.y * dist.y * dist.y),
                               GRAV_CONSTANT * mass[i] * mass[j] / (dist.z * dist.z * dist.z)};
                f.x += mag.x * r_mag.x;
                f.y += mag.y * r_mag.y;
                f.z += mag.z * r_mag.z;
            }
        }
        force[i] = f;
    }
}

__global__ void computeAcceleration(int n, vector3 *pos, vector3 *vel, double *mass, vector3 *force, vector3 *acc) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        vector3 a = {force[i].x / mass[i], force[i].y / mass[i], force[i].z / mass[i]};
        acc[i] = a;
    }
}

void compute(int n, vector3 *pos, vector3 *vel, double *mass, vector3 *acc) {
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    vector3 *d_pos, *d_vel, *d_force, *d_acc;
    double *d_mass;
    hipMalloc(&d_pos, n * sizeof(vector3));
    hipMalloc(&d_vel, n * sizeof(vector3));
    hipMalloc(&d_mass, n * sizeof(double));
    hipMalloc(&d_force, n * sizeof(vector3));
    hipMalloc(&d_acc, n * sizeof(vector3));
    hipMemcpy(d_pos, pos, n * sizeof(vector3), hipMemcpyHostToDevice);
    hipMemcpy(d_vel, vel, n * sizeof(vector3), hipMemcpyHostToDevice);
    hipMemcpy(d_mass, mass, n * sizeof(double), hipMemcpyHostToDevice);
    computeForces<<<numBlocks, BLOCK_SIZE>>>(n, d_pos, d_vel, d_mass, d_force);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in computeForces: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    computeAcceleration<<<numBlocks, BLOCK_SIZE>>>(n, d_pos, d_vel, d_mass, d_force, d_acc);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in computeAcceleration: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipMemcpy(acc, d_acc, n * sizeof(vector3), hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in hipMemcpy: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_mass);
    hipFree(d_force);
    hipFree(d_acc);
}
